#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {

        // Block variables
        int blockSize = 128;
        dim3 threadsPerBlock(blockSize);

        bool MEMORY_BANK_OPTIMIIZED = 1;

        // Data buffers
        int* dev_idata;
        int* dev_bools;
        int* dev_scanned;
        int* dev_indices;
        int* dev_odata;
        int* dev_blockSums;

        // Macros for avoiding shared memory bank conflicts
        #define NUM_BANKS 32
        #define LOG_NUM_BANKS 5
        #define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)

        void setBlockSize(int newBlockSize) {
          blockSize = newBlockSize;
          threadsPerBlock = dim3(blockSize);
        }

        void setMemoryBankOptimized(bool memBankOptimized) {
          MEMORY_BANK_OPTIMIIZED = memBankOptimized;
        }

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kernUpSweep(int nearestPow2, int currOffset, int* data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          int parentIdx = (index + 1) * currOffset * 2 - 1;
          int leftChildIdx = parentIdx - currOffset;
          if (parentIdx >= nearestPow2 || leftChildIdx < 0 || leftChildIdx >= nearestPow2 || parentIdx < 0) {
            return;
          }
          data[parentIdx] += data[leftChildIdx];
        }

        __global__ void kernDownSweep(int nearestPow2, int currOffset, int* data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          int parentIdx = (index + 1) * currOffset * 2 - 1;
          int leftChildIdx = parentIdx - currOffset;
          if (parentIdx >= nearestPow2 || leftChildIdx < 0 || leftChildIdx >= nearestPow2 || parentIdx < 0) {
            return;
          }
          int temp = data[leftChildIdx];
          data[leftChildIdx] = data[parentIdx];
          data[parentIdx] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
            int iters = ilog2ceil(n);
            int nearestPow2 = 1 << iters;

            // set up device arrays to the nearest power of 2
            hipMalloc((void**)&dev_indices, (nearestPow2 + 1) * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemset(dev_indices, 0, (nearestPow2 + 1) * sizeof(int));
            checkCUDAError("hipMemset dev_indices failed!");
            hipMemcpy(dev_indices, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            timer().startGpuTimer();

            // upsweep
            for (int d = 0; d < iters; d++) {
              // calculate power of 2 offset with bitshift
              int currOffset = 1 << d;
              // Only call the number of threads that actually need to write no values in the current sweep level
              dim3 fullBlocksPerGrid((nearestPow2 / (currOffset * 2) + blockSize - 1) / blockSize);
              kernUpSweep<<<fullBlocksPerGrid, threadsPerBlock>>> (nearestPow2, currOffset, dev_indices);
              checkCUDAError("kernUpSweep failed");
              hipError_t e = hipDeviceSynchronize();                       // runtime errors
              if (e != hipSuccess) { fprintf(stderr, "upsweep error: %s\n", hipGetErrorString(e)); }
            }

            // Set last value after upsweep to 0
            hipMemset(dev_indices + nearestPow2 - 1, 0, sizeof(int));
            // downsweep
            for (int d = iters - 1; d >= 0; d--) {
              // calculate power of 2 offset with bitshift
              int currOffset = 1 << d;
              // Only call the number of threads that actually need to write no values in the current sweep level
              dim3 fullBlocksPerGrid((nearestPow2 / (currOffset * 2) + blockSize - 1) / blockSize);
              kernDownSweep<<<fullBlocksPerGrid, threadsPerBlock >>> (nearestPow2, currOffset, dev_indices);
              checkCUDAError("kernDownSweep failed");
              hipError_t e = hipDeviceSynchronize();                       // runtime errors
              if (e != hipSuccess) { fprintf(stderr, "downsweep error: %s\n", hipGetErrorString(e)); }
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy to host failed!");

            // Free device arrays
            hipFree(dev_indices);
            checkCUDAError("hipFree dev_indices failed!");

        }

        // bank conflict unoptimized version
        __global__ void kernSharedMemScan(int n, int* odata, int* idata, int* blockSums) {
          extern __shared__ int temp[];
          // Only one block to maintain shared memory
          int index = threadIdx.x;
          int blockStartIndex = blockIdx.x * 2048;
          int offset = 1;
          //load entire input into shared mem
          temp[2 * index] = idata[2 * index + blockStartIndex];
          temp[2 * index + 1] = idata[2 * index + blockStartIndex + 1];
          // upsweep
          for (int d = n >> 1; d > 0; d >>= 1) {
            __syncthreads();
            if (index < d) {
              int leftChild = offset * (2 * index + 1) - 1;
              int parent = offset * (2 * index + 2) - 1;
              temp[parent] += temp[leftChild];
            }
            offset *= 2;
          }
          __syncthreads();
          // capture last elem in block sums then zero it out zero out last element of temp array
          if (index == 0) {
            blockSums[blockIdx.x] = temp[n - 1];
            temp[n - 1] = 0;
          }
          //downsweep
          for (int d = 1; d < n; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (index < d) {
              int leftChild = offset * (2 * index + 1) - 1;
              int parent = offset * (2 * index + 2) - 1;
              int saved = temp[leftChild];
              temp[leftChild] = temp[parent];
              temp[parent] += saved;
            }
          }
          __syncthreads();
          odata[2 * index + blockStartIndex] = temp[2 * index];
          odata[2 * index + blockStartIndex + 1] = temp[2 * index + 1];

        }

        // bank optimized version
        __global__ void kernSharedMemBankOptimizedScan(int n, int* odata, int* idata, int* blockSums) {
          extern __shared__ int temp[];
          // Only one block to maintain shared memory
          int index = threadIdx.x;
          int blockStartIndex = blockIdx.x * 2048;
          int offset = 1;
          //load entire input into shared mem
          int dataToLoadA = index;
          int dataToLoadB = index + (n / 2);
          int bankOffsetA = CONFLICT_FREE_OFFSET(dataToLoadA);
          int bankOffsetB = CONFLICT_FREE_OFFSET(dataToLoadB);

          temp[dataToLoadA + bankOffsetA] = idata[dataToLoadA + blockStartIndex];
          temp[dataToLoadB + bankOffsetB] = idata[dataToLoadB + blockStartIndex];

          // upsweep
          for (int d = n >> 1; d > 0; d >>= 1) {
            __syncthreads();
            if (index < d) {
              int leftChild = offset * (2 * index + 1) - 1;
              int parent = offset * (2 * index + 2) - 1;
              leftChild += CONFLICT_FREE_OFFSET(leftChild);
              parent += CONFLICT_FREE_OFFSET(parent);
              temp[parent] += temp[leftChild];
            }
            offset *= 2;
          }
          __syncthreads();
          // capture last elem in block sums then zero it out zero out last element of temp array
          if (index == 0) {
            blockSums[blockIdx.x] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
            temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
          }
          //downsweep
          for (int d = 1; d < n; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (index < d) {
              int leftChild = offset * (2 * index + 1) - 1;
              int parent = offset * (2 * index + 2) - 1;
              leftChild += CONFLICT_FREE_OFFSET(leftChild);
              parent += CONFLICT_FREE_OFFSET(parent);
              int saved = temp[leftChild];
              temp[leftChild] = temp[parent];
              temp[parent] += saved;
            }
          }
          __syncthreads();
          odata[dataToLoadA + blockStartIndex] = temp[dataToLoadA + bankOffsetA];
          odata[dataToLoadB + blockStartIndex] = temp[dataToLoadB + bankOffsetB];
          
        }

        __global__ void kernAddBlockSums(int n, int* odata, int* blockSums) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index >= n) {
            return;
          }
          int blockSumIndex = index / 2048;
          odata[index] += blockSums[blockSumIndex];
        }

        void sharedMemScan(int n, int* odata, const int* idata) {
          // max allowed in shared memory of one block
          if (n > 1 << 22) {
            timer().startGpuTimer();
            timer().endGpuTimer();
            return;
          }
          int iters = ilog2ceil(n);
          int nearestPow2 = 1 << iters;

          // each individual block can handle 2048 data points
          const int SPLIT = 2048;

          // set up device arrays to the nearest power of 2
          hipMalloc((void**)&dev_indices, nearestPow2 * sizeof(int));
          checkCUDAError("hipMalloc dev_indices failed!");
          hipMalloc((void**)&dev_odata, nearestPow2 * sizeof(int));
          checkCUDAError("hipMalloc dev_odata failed!");
          hipMalloc((void**)&dev_blockSums, (((nearestPow2 + SPLIT - 1) / SPLIT)) * sizeof(int));
          checkCUDAError("hipMalloc dev_blockSums failed!");
          hipMalloc((void**)&dev_scanned, (((nearestPow2 + SPLIT - 1) / SPLIT)) * sizeof(int));
          checkCUDAError("hipMalloc dev_scanned failed!");

          hipMemset(dev_indices, 0, nearestPow2 * sizeof(int));
          checkCUDAError("hipMemset dev_indices failed!");
          hipMemcpy(dev_indices, idata, n * sizeof(int), hipMemcpyHostToDevice);
          checkCUDAError("hipMemcpy to device failed!");

          timer().startGpuTimer();
          
          const int maxThreadsPerBlock = 1024;
          int scannedSoFar = 0;
          int blocksNeeded = (nearestPow2 + 2048 - 1) / 2048;

          if (blocksNeeded == 1) {
            // only need one block
            if (MEMORY_BANK_OPTIMIIZED) {
              kernSharedMemBankOptimizedScan << <1, nearestPow2 / 2, (nearestPow2 + CONFLICT_FREE_OFFSET(nearestPow2))* sizeof(int) >> > (nearestPow2, dev_odata, dev_indices, dev_blockSums);
            }
            else {
              kernSharedMemScan << <1, nearestPow2 / 2, nearestPow2 * sizeof(int) >> > (nearestPow2, dev_odata, dev_indices, dev_blockSums);
            }
          }
          else {
            // need multiple blocks and to scan block sums

            if (MEMORY_BANK_OPTIMIIZED) {
              kernSharedMemBankOptimizedScan<<<blocksNeeded, 1024,( 2048 + (CONFLICT_FREE_OFFSET(2048))) * sizeof(int) >>> (2048, dev_odata, dev_indices, dev_blockSums);
              kernSharedMemBankOptimizedScan<<<1, (blocksNeeded + 1) / 2, (blocksNeeded + CONFLICT_FREE_OFFSET(blocksNeeded)) * sizeof(int) >>> (blocksNeeded, dev_scanned, dev_blockSums, dev_blockSums);

              dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
              kernAddBlockSums<<<fullBlocksPerGrid, threadsPerBlock >> > (n, dev_odata, dev_scanned);
            }
            else {
              kernSharedMemScan<<<blocksNeeded, 1024, 1024 * 2 * sizeof(int) >>> (2048, dev_odata, dev_indices, dev_blockSums);
              kernSharedMemScan<<< 1, (blocksNeeded + 1) / 2, blocksNeeded * sizeof(int) >>> (blocksNeeded, dev_scanned, dev_blockSums, dev_blockSums);

              dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
              kernAddBlockSums<<<fullBlocksPerGrid, threadsPerBlock >>> (n, dev_odata, dev_scanned);
            }
            
          }

          timer().endGpuTimer();

          // copy data back over
          hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
          checkCUDAError("hipMemcpy to host failed!");

          // free
          hipFree(dev_indices);
          hipFree(dev_odata);
          hipFree(dev_blockSums);
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // TODO

            int iters = ilog2ceil(n);
            int nearestPow2 = 1 << iters;
            // set up device arrays to the nearest power of 2
            hipMalloc((void**)&dev_bools, nearestPow2 * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_idata, nearestPow2 * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_indices, nearestPow2 * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_odata, nearestPow2 * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemset(dev_indices, 0, nearestPow2 * sizeof(int));
            checkCUDAError("hipMemset dev_indices failed!");
            hipMemset(dev_bools, 0, nearestPow2 * sizeof(int));
            checkCUDAError("hipMemset dev_bools failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            timer().startGpuTimer();

            // map
            dim3 fullBlocksPerGrid((nearestPow2 + blockSize - 1) / blockSize);
            Common::kernMapToBoolean<<<(n + blockSize - 1) / blockSize, threadsPerBlock>>> (n, dev_bools, dev_idata);

            // scan
            
            // Copy the bools to the indices array for scan
            hipMemcpy(dev_indices, dev_bools, nearestPow2 * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_bools to dev_indices failed!");

            // upsweep
            for (int d = 0; d < iters; d++) {
              // calculate power of 2 offset with bitshift
              int currOffset = 1 << d;
              // Only call the number of threads that actually need to write no values in the current sweep level
              dim3 fullBlocksPerGrid((nearestPow2 / (currOffset * 2) + blockSize - 1) / blockSize);
              kernUpSweep<<<fullBlocksPerGrid, threadsPerBlock>>> (nearestPow2, currOffset, dev_indices);
            }

            // Set last value after upsweep to 0
            hipMemset(dev_indices + nearestPow2 - 1, 0, sizeof(int));

            //downsweep
            for (int d = iters - 1; d >= 0; d--) {
              // calculate power of 2 offset with bitshift
              int currOffset = 1 << d;
              // Only call the number of threads that actually need to write no values in the current sweep level
              dim3 fullBlocksPerGrid((nearestPow2 / (currOffset * 2) + blockSize - 1) / blockSize);
              kernDownSweep<<<fullBlocksPerGrid, threadsPerBlock>>> (nearestPow2, currOffset, dev_indices);
            }

            // scatter
            Common::kernScatter<<<(n + blockSize - 1) / blockSize, threadsPerBlock >>> (n, dev_odata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();

            // figure out num elements
            int lastIndex;
            int lastBool;
            hipMemcpy(&lastIndex, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastBool, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int size = lastIndex + lastBool;
            hipMemcpy(odata, dev_odata, size * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy to host failed!");

            // Free device arrays
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed!");
            hipFree(dev_bools);
            checkCUDAError("hipFree dev_bools failed!");
            hipFree(dev_indices);
            checkCUDAError("hipFree dev_indices failed!");
            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed!");

            return size;
        }
    }
}
